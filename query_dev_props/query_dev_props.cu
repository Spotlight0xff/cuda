#include <stdio.h>

int main() {
    int nDevices;

    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(err));
        exit(1);
    }

    for (int i=0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        // query the device properties of the i-th device
        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(err));
            exit(1);
        }
        
        printf("Device Number: %d\n", i);
        printf("\tDevice Name: %s\n", prop.name);
        printf("\tMajor compute capability: %d.%d\n", prop.major, prop.minor);
        printf("\tDevice Global Memory: %f GB\n", prop.totalGlobalMem / (1024.0*1024.0*1024.0));
        printf("\tShared Memory per Block: %d bytes\n", prop.sharedMemPerBlock);
        printf("\tMap Host Memory available (pinned Memory): %s\n", prop.canMapHostMemory ? "true": "false");
        printf("\tMemory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("\tMemory Bus Width: %d bits\n", prop.memoryBusWidth);
        printf("\tPeak Memory Bandwidth: %f GB/s\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8)/1.0e6);
        printf("\tNumber of asynchronous engines: %d\n", prop.asyncEngineCount);
        printf("\tL2 Cache bytes: %d\n", prop.l2CacheSize);
        printf("\tConcurrent Kernels: %d\n", prop.concurrentKernels);
    }
}
